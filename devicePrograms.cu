#include "hip/hip_runtime.h"


#include "renderParams.h"
#include "sutil\vec_math.h"

#include "random.h"

#include <crt\host_defines.h>
#include <optix_device.h>
#include "helpers.h"


extern "C" __constant__ RenderParams renderParams;

struct TraceResult {
    int missed = 0;

    float3 position;
    float3 normal;

    Material material;
};


static __forceinline__ __device__
void* UnpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  PackPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* GetPerRayData()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(UnpackPointer(u0, u1));
}

static __forceinline__ __device__ 
float3 RandomInUnitSphere(unsigned int& seed) {
    while (true) {
        float3 v = make_float3(rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f);
        if (dot(v, v) >= 1.0f) continue;
        return v;
    }
}

static __forceinline__ __device__ 
float3 RandomSampleHemisphere(unsigned int& seed, const float3& normal) {
    const float3 vec_in_sphere = RandomInUnitSphere(seed);
    if (dot(vec_in_sphere, normal) > 0.0f)
        return vec_in_sphere;
    else
        return -vec_in_sphere;
}

static __forceinline__ __device__
void RayTrace(float3 position, float3 rayDir, int rayType, TraceResult* result) {
    uint32_t u0, u1;
    PackPointer(result, u0, u1);
    optixTrace(
        renderParams.traversable,
        position,
        rayDir,
        0.001, 1e20f, 0,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        rayType,
        RAY_TYPE_COUNT,
        rayType, u0, u1);
}


extern "C" __global__ void __raygen__renderFrame() {
    const int px = optixGetLaunchIndex().x;
    const int py = optixGetLaunchIndex().y;

    const auto& camera = renderParams.camera;

    unsigned int seed = tea<4>(px + py * renderParams.screenSize.x, renderParams.frame.frameId);

    float3 result = make_float3(0);

    TraceResult traceResult;
    

    for (int i = 0; i < renderParams.samplesPerLaunch; i++) {
        const float2 screenPos = make_float2(px + rnd(seed), py + rnd(seed)) / make_float2(renderParams.screenSize);

        float3 rayOrigin = camera.position;
        float3 rayDir = normalize(camera.direction + (screenPos.y - 0.5f) * camera.vertical + (screenPos.x - 0.5f) * camera.horizontal);

        // brdf * cos / pdf(input)
        float3 attenuation = make_float3(1.f);

        for (int depth = 0; depth < renderParams.maxDepth; depth++) {
            traceResult.missed = true;

            if (rnd(seed) > renderParams.russianRouletteProbability) {
                break;
            }

            RayTrace(rayOrigin, rayDir, RADIANCE_RAY_TYPE, &traceResult);

            if (traceResult.missed) {
                break;
            }

            result += traceResult.material.emission * attenuation;

            attenuation *= traceResult.material.albedo / renderParams.russianRouletteProbability;

            rayOrigin = traceResult.position;
            rayDir = RandomSampleHemisphere(seed, traceResult.normal);

        }

    }

    result /= renderParams.samplesPerLaunch;

    const uint32_t colorBufferIndex = (renderParams.screenSize.x - px - 1) + (renderParams.screenSize.y - py - 1) * renderParams.screenSize.x;

    const int subframeCount = renderParams.frame.subframeCount;

    if (subframeCount > 0) {
        result = lerp(renderParams.frame.colorBuffer[colorBufferIndex], result, 1. / subframeCount);
    }

    // const uint32_t colorBufferIndex = px + py * renderParams.screenSize.x;
    renderParams.frame.colorBuffer[colorBufferIndex] = result;
}

extern "C" __global__ void __miss__radiance() {

}

extern "C" __global__ void __closesthit__mesh() { 
    TraceResult &result = *GetPerRayData<TraceResult>();

    const ShaderBindingData& sbtData = *(const ShaderBindingData*)optixGetSbtDataPointer();
    const Material& material = sbtData.material;
    const DeviceMeshData& data = sbtData.data.mesh;

    const int primID = optixGetPrimitiveIndex();
    const int3 index = data.index[primID];
    const float3& A = data.vertex[index.x];
    const float3& B = data.vertex[index.y];
    const float3& C = data.vertex[index.z];
    const float3 normal = normalize(cross(B - A, C - A));
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();

    const float3 rayDir = optixGetWorldRayDirection();

    result.missed = false;
    result.normal = -normal;
    result.position = position;
    result.material = material;
}


extern "C" __global__ void __closesthit__sphere() {
    TraceResult& result = *GetPerRayData<TraceResult>();

    const ShaderBindingData& sbtData = *(const ShaderBindingData*)optixGetSbtDataPointer();
    const Material& material = sbtData.material;
    const DeviceSphereData& data = sbtData.data.sphere;

    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    const float3 normal = normalize(position - data.position);

    const float3 rayDir = optixGetWorldRayDirection();

    result.missed = false;
    result.normal = normal;
    result.position = position;
    result.material = material;    
}
