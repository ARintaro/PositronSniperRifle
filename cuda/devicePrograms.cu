#include "hip/hip_runtime.h"


#include "renderParams.h"
#include "random.h"
#include "disney.cuh"
#include "bezier.cuh"
#include "material.cuh"
#include "deviceHelper.cuh"


static __forceinline__ __device__
bool ApplyAirScatter(float3& result, PathState& state, TraceResult& traceResult) {
    if (renderParams.globalFogDensity <= 0) return false;

    float scatterProbability = 1 - expf(-renderParams.globalFogDensity * traceResult.distance);
    // float scatterProbability = min(renderParams.globalFogDensity * distance, 1.f);
    float sampledScatter = rnd(state.seed);

    if (sampledScatter < scatterProbability) {
        // ����ɢ��
        float scateredDistance = -(1.f / renderParams.globalFogDensity) * logf(1 - sampledScatter);

        // ����trace���
        traceResult.directLightId = -1;
        traceResult.distance = scateredDistance;
        traceResult.position = state.rayOrigin + scateredDistance * state.rayDir;
        traceResult.normal = RandomInUnitSphere(state.seed);

        SAMPLE_DIRECT_LIGHT(result, state, traceResult, renderParams.globalFogAttenuation / scatterProbability);

        state.rayOrigin = traceResult.position;
        state.rayDir = reflect(state.rayDir, traceResult.normal);
        state.attenuation *= renderParams.globalFogAttenuation / renderParams.russianRouletteProbability;
        return true;
    }

    return false;
}

extern "C" __global__ void __raygen__renderFrame() {
    const int px = optixGetLaunchIndex().x;
    const int py = optixGetLaunchIndex().y;

    const auto& camera = renderParams.camera;

    unsigned int seed = tea<4>(px + py * renderParams.screenSize.x, renderParams.frame.frameId);

    float3 result = make_float3(0);

    TraceResult traceResult;

    for (int i = 0; i < renderParams.samplesPerLaunch; i++) {
        const float2 screenPos = make_float2(px + rnd(seed), py + rnd(seed)) / make_float2(renderParams.screenSize);

        float2 unitDisk = camera.lenRadius * RandomInUnitDisk(seed);


        PathState state;
        state.seed = seed;
        state.rayOrigin = camera.position + unitDisk.x * normalize(camera.horizontal) + unitDisk.y * normalize(camera.vertical);
        state.rayDir = normalize(camera.direction + (screenPos.y - 0.5f) * camera.vertical + (screenPos.x - 0.5f) * camera.horizontal);
        state.attenuation = make_float3(1.f);
        state.supposedColor = make_float3(1.f);
        state.collectDirectLight = true;
       

        for (int depth = 0; depth < renderParams.maxDepth; depth++) {
            traceResult.missed = true;
            traceResult.distance = 1e9;

            if (rnd(seed) > renderParams.russianRouletteProbability) {
                break;
            }

            RayTrace(state.rayOrigin, state.rayDir, RADIANCE_RAY_TYPE, &traceResult);

            if (ApplyAirScatter(result, state, traceResult)) continue;

            if (traceResult.missed) break;
  
            optixDirectCall<void, float3&, PathState&, TraceResult&>(traceResult.material.programIndex, result, state, traceResult);

            if (traceResult.missed) break;
        }

    }

    result /= renderParams.samplesPerLaunch;
    const uint32_t colorBufferIndex = (renderParams.screenSize.x - px - 1) + (renderParams.screenSize.y - py - 1) * renderParams.screenSize.x;
    const int subframeCount = renderParams.frame.subframeCount;

    if (isnan(result.x)) result.x = 0;
    if (isnan(result.y)) result.y = 0;
    if (isnan(result.z)) result.z = 0;

    if (subframeCount > 0) {
        result = lerp(renderParams.frame.colorBuffer[colorBufferIndex], result, 1. / subframeCount);
    }

    renderParams.frame.colorBuffer[colorBufferIndex] = result;
}

extern "C" __global__ void __miss__radiance() {

}

extern "C" __global__ void __closesthit__mesh() { 
    TraceResult &result = *GetPerRayData<TraceResult>();

    const ShaderBindingData& sbtData = *(const ShaderBindingData*)optixGetSbtDataPointer();
    const Material& material = sbtData.material;
    const DeviceMeshData& data = sbtData.data.mesh;

    const int primID = optixGetPrimitiveIndex();
    const int3 index = data.index[primID];
    float3 normal = make_float3(0);
    float2 bary = optixGetTriangleBarycentrics();
    float2 texcoord = make_float2(0);
    float3 tangent = make_float3(0);

    if (data.normal) {
        normal = (1.f - bary.x - bary.y) * data.normal[index.x] + bary.x * data.normal[index.y] + bary.y * data.normal[index.z];
    } else {
        const float3& A = data.vertex[index.x];
        const float3& B = data.vertex[index.y];
        const float3& C = data.vertex[index.z];
        normal = normalize(cross(B - A, C - A));
    }

    if (data.texcoord) {
        texcoord = (1.f - bary.x - bary.y) * data.texcoord[index.x] + bary.x * data.texcoord[index.y] + bary.y * data.texcoord[index.z];

        {
            // tangent
            float3 e1 = data.vertex[index.y] - data.vertex[index.x], e2 = data.vertex[index.z] - data.vertex[index.x];
            float2 duv1 = data.texcoord[index.y] - data.texcoord[index.x], duv2 = data.texcoord[index.z] - data.texcoord[index.x];
            tangent = (duv1.y * e2 - duv2.y * e1) / (duv1.y * duv2.x - duv2.y * duv1.x);
            tangent = normalize(tangent - dot(tangent, normal) * normal);
        }
    }

    

    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    const float3 rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, normal) > 0) {
        normal = -normal;
        result.outer = false;
    }

    result.missed = false;
    result.normal = normal;
    result.position = position;
    result.material = material;
    result.distance = optixGetRayTmax();
    result.directLightId = sbtData.directLightId;
    result.texcoord = texcoord;
    result.tangent = tangent;
}


extern "C" __global__ void __closesthit__sphere() {
    TraceResult& result = *GetPerRayData<TraceResult>();

    const ShaderBindingData& sbtData = *(const ShaderBindingData*)optixGetSbtDataPointer();
    const Material& material = sbtData.material;
    const DeviceSphereData& data = sbtData.data.sphere;

    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    float3 normal = normalize(position - data.position);

    const float3 rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, normal) > 0) {
        normal = -normal;
        result.outer = false;
    }

    result.missed = false;
    result.normal = normal;
    result.position = position;
    result.material = material;    
    result.distance = optixGetRayTmax();
    result.directLightId = -1;
    // TODO : texcrood

    // TODO : tangent
}

